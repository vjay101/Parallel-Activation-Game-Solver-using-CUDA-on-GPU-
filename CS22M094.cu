#include "hip/hip_runtime.h"
/*
 * Title: CS6023, GPU Programming, Jan-May 2023, Assignment-3
 * Description: Activation Game 
 */

#include <cstdio>        // Added for printf() function 
#include <sys/time.h>    // Added to get time of day
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
#include "graph.hpp"
 
using namespace std;


ofstream outfile; // The handle for printing the output

/******************************Write your kerenels here ************************************/


//This kernel is used to find level 0 nodes

__global__ void initializeLevel0(int currLevel, int *d_offset, int *d_csrList, int *d_vertexLevel, int level0lastnode){
    int id=blockIdx.x*blockDim.x+threadIdx.x;
   
    if(id <= level0lastnode){
        d_vertexLevel[id]=0;
        for(int j = d_offset[id]; j<d_offset[id+1]; j++){
            atomicCAS(&d_vertexLevel[d_csrList[j]], -1, currLevel+1);
           
        }
    }
}

//this kernel will find the nodes of corresponding level from 1 to L-1

__global__ void find_level(int currLevel,int *d_offset, int *d_csrList, int *d_vertexLevel){
    int id=blockIdx.x*blockDim.x+threadIdx.x; 
    
    if(d_vertexLevel[id]==currLevel){
        for(int j = d_offset[id]; j<d_offset[id+1]; j++){
           atomicCAS(&d_vertexLevel[d_csrList[j]], -1, currLevel+1);
        }
    }
  

}

//this kernel will activate level no 0 and at same time update aid of level 1
__global__ void zerolevelactive(int *d_offset,int *d_csrList,int *d_apr,int *d_aid,int *d_activeVertex,int *d_isactiveNode,int *d_levelfind,int L,int V,int E){
 
 int id=blockIdx.x*blockDim.x+threadIdx.x;
  if(id <= d_levelfind[0]){
   
        d_isactiveNode[id]=1;
        
        for(int j=d_offset[id];j<d_offset[id+1];j++){
           atomicAdd(&d_aid[d_csrList[j]], 1);
        }
    }
}

//this kernel will activate from level no 1 to last level
  __global__ void activation(int count,int *d_offset,int *d_csrList,int *d_apr,int *d_aid,int *d_activeVertex,int *d_isactiveNode,int *d_levelfind,int L,int V,int E){
    
       int id=blockIdx.x*blockDim.x+threadIdx.x;
       id= id + d_levelfind[count]+1;
       if(id <= d_levelfind[count+1]){
        if(d_aid[id]>=d_apr[id]){
          d_isactiveNode[id]=1;
         
          for(int j=d_offset[id];j<d_offset[id+1];j++){
              atomicAdd(&d_aid[d_csrList[j]], 1);

           }
        }
       }
}

//this kernel will deactivate if a node doesnot satisfy the condition to remain active
__global__ void de_activation(int count,int *d_offset,int *d_csrList,int *d_apr,int *d_aid,int *d_activeVertex,int *d_isactiveNode,int *d_levelfind,int L,int V,int E){
    
         int id=blockIdx.x*blockDim.x+threadIdx.x;
          id= id + d_levelfind[count]+1;  
          if(id <= d_levelfind[count+1]){ 
           if(id>d_levelfind[count]+1 && id<d_levelfind[count+1] && d_isactiveNode[id]==1 && d_isactiveNode[id-1]==0 && d_isactiveNode[id+1]==0){
           d_isactiveNode[id]=0;
          
           for(int j=d_offset[id];j<d_offset[id+1];j++){
              atomicAdd(&d_aid[d_csrList[j]],-1);
           }
        }
    }

    
}


//this kernel will find the final ans in which all the nodes V will run parallel
__global__ void active_vertex(int *d_vertexLevel,int *d_offset,int *d_csrList,int *d_apr,int *d_aid,int *d_activeVertex,int *d_isactiveNode,int *d_levelfind,int L,int V,int E){
    
  
    int id=blockIdx.x*blockDim.x+threadIdx.x;
     if(id<V){
         if(d_isactiveNode[id]==1)
         {
             atomicAdd(&d_activeVertex[d_vertexLevel[id]], 1);
         }
    }
    
  
}
   
     
    
    
    
/**************************************END*************************************************/



//Function to write result in output file
void printResult(int *arr, int V,  char* filename){
    outfile.open(filename);
    for(long int i = 0; i < V; i++){
        outfile<<arr[i]<<" ";   
    }
    outfile.close();
}

/**
 * Timing functions taken from the matrix multiplication source code
 * rtclock - Returns the time of the day 
 * printtime - Prints the time taken for computation 
 **/
double rtclock(){
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d", stat);
    return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime){
    printf("%s%3f seconds\n", str, endtime - starttime);
}

int main(int argc,char **argv){
    // Variable declarations
    int V ; // Number of vertices in the graph
    int E; // Number of edges in the graph
    int L; // number of levels in the graph

    //Reading input graph
    char *inputFilePath = argv[1];
    graph g(inputFilePath);

    //Parsing the graph to create csr list
    g.parseGraph();

    //Reading graph info 
    V = g.num_nodes();
    E = g.num_edges();
    L = g.get_level();


    //Variable for CSR format on host
    int *h_offset; // for csr offset
    int *h_csrList; // for csr
    int *h_apr; // active point requirement
    

    //reading csr
    h_offset = g.get_offset();
    h_csrList = g.get_csr();   
    h_apr = g.get_aprArray();
    
    // Variables for CSR on device
    int *d_offset;
    int *d_csrList;
    int *d_apr; //activation point requirement array
    int *d_aid; // active in-degree array
    //Allocating memory on device 
    hipMalloc(&d_offset, (V+1)*sizeof(int));
    hipMalloc(&d_csrList, E*sizeof(int)); 
    hipMalloc(&d_apr, V*sizeof(int)); 
    hipMalloc(&d_aid, V*sizeof(int));

    //copy the csr offset, csrlist and apr array to device
    hipMemcpy(d_offset, h_offset, (V+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrList, h_csrList, E*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_apr, h_apr, V*sizeof(int), hipMemcpyHostToDevice);

    
/***Important***/

// Initialize d_aid array to zero for each vertex
// Make sure to use comments

/***END***/
double starttime = rtclock(); 

/*********************************CODE AREA*****************************************/
// variable for result, storing number of active vertices at each level, on host
    int *h_activeVertex;
    h_activeVertex = (int*)malloc(L*sizeof(int));
    // setting initially all to zero
    memset(h_activeVertex, 0, L*sizeof(int));

    // variable for result, storing number of active vertices at each level, on device
    int *d_activeVertex;
    hipMalloc(&d_activeVertex, L*sizeof(int));
    //hipMemcpy(d_activeVertex, h_activeVertex, L*sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_activeVertex, 0, L*sizeof(int));
   

    int *h_aid;
    h_aid = (int *)malloc(V*sizeof(int));
    // setting initially all to zero
    memset(h_aid, 0, V*sizeof(int));

   
    hipMalloc(&d_aid, V*sizeof(int));
    //hipMemcpy(d_aid, h_aid, V*sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_aid, 0, V*sizeof(int));
   
    int *h_isactiveNode;
    h_isactiveNode = (int*)malloc(V*sizeof(int));
    // setting initially all to zero
    memset(h_isactiveNode, 0, V*sizeof(int)); 

    int *d_isactiveNode;
    hipMalloc(&d_isactiveNode, V*sizeof(int));
    
    hipMemset(d_isactiveNode, 0, V*sizeof(int));

    int *h_levelfind;
    h_levelfind = (int*)malloc(L*sizeof(int));
    // setting initially all to zero
    memset(h_levelfind, 0, L*sizeof(int)); 

    int *d_levelfind;
    hipMalloc(&d_levelfind, L*sizeof(int));
    
    hipMemset(d_levelfind,0,L*sizeof(int));

    int *h_vertexLevel;
    h_vertexLevel = (int*)malloc(V*sizeof(int));
    int *d_vertexLevel;
    hipMalloc(&d_vertexLevel, V*sizeof(int));
    hipMemset(d_vertexLevel,-1,V*sizeof(int));



//launching of kernel threads here
//calculating the level no of each node
    
    // Level 0 ka last nikal le 

    // numThread = Level 0 pe nodes hai 
    int lasLevel0Node; 
    for(int i=0;h_apr[i]==0;i++){
      lasLevel0Node=i;
    }
     
    //kernel ko launch krna 
    int numBlock = (V+1024-1)/1024;
    int numThread = 1024;
    initializeLevel0<<<numBlock, numThread>>>(0, d_offset, d_csrList, d_vertexLevel, lasLevel0Node);
    hipMemcpy(h_vertexLevel,d_vertexLevel,V*sizeof(int),hipMemcpyDeviceToHost);

    //for finding the level of nodes
    for(int i=1;i<L-1;i++){
        find_level<<<numBlock, numThread>>>(i,d_offset, d_csrList, d_vertexLevel);
     }
    hipMemcpy(h_vertexLevel,d_vertexLevel,V*sizeof(int),hipMemcpyDeviceToHost);
   //calculating end edges from above level node array
   int l=0;
   for(int i=0;i<V-1;i++){
      if(h_vertexLevel[i]!=h_vertexLevel[i+1])
        {
            h_levelfind[l]=i;
            l++;
        }
   }
    h_levelfind[L-1]=V-1; 
    hipMemcpy(d_levelfind,h_levelfind,L*sizeof(int),hipMemcpyHostToDevice);
  
    
    //kernel:checking activeness of nodes of level 0 and at same time update aid of level 1

     zerolevelactive<<<(h_levelfind[0]+1024)/1024,1024>>>(d_offset,d_csrList,d_apr,d_aid,d_activeVertex,d_isactiveNode,d_levelfind,L,V,E);
     hipMemcpy(h_aid,d_aid,V*sizeof(int),hipMemcpyDeviceToHost);
     hipMemcpy(h_isactiveNode,d_isactiveNode,V*sizeof(int),hipMemcpyDeviceToHost);


    //kernel:checking activeness of L1 to last level
    
    for(int i=1;i<L;i++){
        int num_block = (h_levelfind[i]-h_levelfind[i-1]+1024)/1024;
        activation<<<num_block, 1024>>>(i-1,d_offset,d_csrList,d_apr,d_aid,d_activeVertex,d_isactiveNode,d_levelfind,L,V,E);
        hipDeviceSynchronize();
        de_activation<<<num_block, 1024>>>(i-1,d_offset,d_csrList,d_apr,d_aid,d_activeVertex,d_isactiveNode,d_levelfind,L,V,E);
        hipDeviceSynchronize();
       
    }
     hipMemcpy(h_aid,d_aid,V*sizeof(int),hipMemcpyDeviceToHost);
   
    

    //kernel:to calculate the final result

    active_vertex<<<(V+1024)/1024,1024>>>(d_vertexLevel,d_offset,d_csrList,d_apr,d_aid,d_activeVertex,d_isactiveNode,d_levelfind,L,V,E);
    hipMemcpy(h_activeVertex,d_activeVertex,L*sizeof(int),hipMemcpyDeviceToHost);
 //finally the result is in h_activeVertex
   

/********************************END OF CODE AREA**********************************/
double endtime = rtclock();  
printtime("GPU Kernel time: ", starttime, endtime);  

// --> Copy C from Device to Host
hipMemcpy(h_activeVertex, d_activeVertex, L*sizeof(int), hipMemcpyDeviceToHost);
char outFIle[30] = "./output.txt" ;
printResult(h_activeVertex, L, outFIle);
if(argc>2)
{
    for(int i=0; i<L; i++)
    {
        printf("level = %d , active nodes = %d\n",i,h_activeVertex[i]);
    }
}
}